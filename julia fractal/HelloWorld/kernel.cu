#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstring>
#include <GL\glut.h>
using namespace std;
//ERRORS
static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		cout << hipGetErrorString(err) << "in" << file << "at" << line << "line" << endl;
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//CPUBitmap
struct CPUBitmap {
	unsigned char    *pixels;
	int     x, y;
	void    *dataBlock;
	void(*bitmapExit)(void*);

	CPUBitmap(int width, int height, void *d = NULL) {
		pixels = new unsigned char[width * height * 4];
		x = width;
		y = height;
		dataBlock = d;
	}

	~CPUBitmap() {
		delete[] pixels;
	}

	unsigned char* get_ptr(void) const { return pixels; }
	long image_size(void) const { return x * y * 4; }

	void display_and_exit(void(*e)(void*) = NULL) {
		CPUBitmap**   bitmap = get_bitmap_ptr();
		*bitmap = this;
		bitmapExit = e;
		// a bug in the Windows GLUT implementation prevents us from
		// passing zero arguments to glutInit()
		int c = 1;
		char* dummy = "";
		glutInit(&c, &dummy);
		glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);
		glutInitWindowSize(x, y);
		glutCreateWindow("bitmap");
		glutKeyboardFunc(Key);
		glutDisplayFunc(Draw);
		glutMainLoop();
	}

	// static method used for glut callbacks
	static CPUBitmap** get_bitmap_ptr(void) {
		static CPUBitmap   *gBitmap;
		return &gBitmap;
	}

	// static method used for glut callbacks
	static void Key(unsigned char key, int x, int y) {
		switch (key) {
		case 27:
			CPUBitmap * bitmap = *(get_bitmap_ptr());
			if (bitmap->dataBlock != NULL && bitmap->bitmapExit != NULL)
				bitmap->bitmapExit(bitmap->dataBlock);
			exit(0);
		}
	}

	// static method used for glut callbacks
	static void Draw(void) {
		CPUBitmap*   bitmap = *(get_bitmap_ptr());
		glClearColor(0.0, 0.0, 0.0, 1.0);
		glClear(GL_COLOR_BUFFER_BIT);
		glDrawPixels(bitmap->x, bitmap->y, GL_RGBA, GL_UNSIGNED_BYTE, bitmap->pixels);
		glFlush();
	}
};
#define DIM 800
struct  hipComplex
{
	float r;
	float i;
	__device__ hipComplex(float a, float b) : r(a), i(b) {
	//	r = a;
	//	i = b;
	}
	__device__ float magnitude2() {
		return r * r + i * i;
	}
	__device__ hipComplex operator * (const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ hipComplex operator + (const hipComplex& a) {
		return hipComplex(r + a.r, i + a.i);
	}
};
__device__ int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x)/(DIM / 2);
	float jy = scale * (float)(DIM / 2 - y)/(DIM / 2);
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);
	int i = 0;
	for (i = 0; i < 200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}
__global__ void kernel(unsigned char * ptr) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	int juliaValue = julia(x, y);
	ptr[offset * 4 + 0] = 255 * juliaValue;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}
int main()
{	
	CPUBitmap bitmap(DIM, DIM);
	unsigned char * dev_bitmap = bitmap.get_ptr();
	hipMalloc((void**)&dev_bitmap, bitmap.image_size());
	dim3 grid(DIM, DIM);
	kernel<<<grid,1>>>(dev_bitmap);
	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
	bitmap.display_and_exit();
	hipFree(dev_bitmap);
    return 0;
}